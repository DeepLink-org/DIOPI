#include "hip/hip_runtime.h"
/**
 * @file
 * @author DeepLink
 * @copyright  (c) 2023, DeepLink.
 */

#include <float.h>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/library.h>

#include "../cuda_helpers.h"

namespace mmcv {
namespace ops {

using namespace at;

int const threadsPerBlock = sizeof(unsigned long long int) * 8;

__device__ inline bool devIoU(float const *const a, float const *const b,
                              const int offset, const float threshold) {
  float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
  float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
  float width = fmaxf(right - left + offset, 0.f),
        height = fmaxf(bottom - top + offset, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + offset) * (a[3] - a[1] + offset);
  float Sb = (b[2] - b[0] + offset) * (b[3] - b[1] + offset);
  return interS > threshold * (Sa + Sb - interS);
}

__global__ static void nms_cuda(const int n_boxes, const float iou_threshold,
                                const int offset, const float *dev_boxes,
                                unsigned long long *dev_mask) {
  int blocks = (n_boxes + threadsPerBlock - 1) / threadsPerBlock;
  CUDA_2D_KERNEL_BLOCK_LOOP(col_start, blocks, row_start, blocks) {
    const int tid = threadIdx.x;

    if (row_start > col_start) return;

    const int row_size =
        fminf(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
    const int col_size =
        fminf(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

    __shared__ float block_boxes[threadsPerBlock * 4];
    if (tid < col_size) {
      block_boxes[tid * 4 + 0] =
          dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 0];
      block_boxes[tid * 4 + 1] =
          dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 1];
      block_boxes[tid * 4 + 2] =
          dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 2];
      block_boxes[tid * 4 + 3] =
          dev_boxes[(threadsPerBlock * col_start + tid) * 4 + 3];
    }
    __syncthreads();

    if (tid < row_size) {
      const int cur_box_idx = threadsPerBlock * row_start + tid;
      const float *cur_box = dev_boxes + cur_box_idx * 4;
      int i = 0;
      unsigned long long int t = 0;
      int start = 0;
      if (row_start == col_start) {
        start = tid + 1;
      }
      for (i = start; i < col_size; i++) {
        if (devIoU(cur_box, block_boxes + i * 4, offset, iou_threshold)) {
          t |= 1ULL << i;
        }
      }
      dev_mask[cur_box_idx * gridDim.y + col_start] = t;
    }
  }
}

__global__ static void gather_keep_from_mask(bool *keep,
                                             const unsigned long long *dev_mask,
                                             const int n_boxes) {
  const int col_blocks = (n_boxes + threadsPerBlock - 1) / threadsPerBlock;
  const int tid = threadIdx.x;

  // mark the bboxes which have been removed.
  extern __shared__ unsigned long long removed[];

  // initialize removed.
  for (int i = tid; i < col_blocks; i += blockDim.x) {
    removed[i] = 0;
  }
  __syncthreads();

  for (int nblock = 0; nblock < col_blocks; ++nblock) {
    auto removed_val = removed[nblock];
    __syncthreads();
    const int i_offset = nblock * threadsPerBlock;
#pragma unroll
    for (int inblock = 0; inblock < threadsPerBlock; ++inblock) {
      const int i = i_offset + inblock;
      if (i >= n_boxes) break;
      // select a candidate, check if it should kept.
      if (!(removed_val & (1ULL << inblock))) {
        if (tid == 0) {
          // mark the output.
          keep[i] = true;
        }
        auto p = dev_mask + i * col_blocks;
        // remove all bboxes which overlap the candidate.
        for (int j = tid; j < col_blocks; j += blockDim.x) {
          if (j >= nblock) removed[j] |= p[j];
        }
        __syncthreads();
        removed_val = removed[nblock];
      }
    }
  }
}

Tensor NMSCUDAKernelLauncher(Tensor boxes, Tensor scores, float iou_threshold,
                             int offset) {
  at::cuda::CUDAGuard device_guard(boxes.device());

  if (boxes.numel() == 0) {
    return at::empty({0}, boxes.options().dtype(at::kLong));
  }
  auto order_t = std::get<1>(scores.sort(0, /*descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);
  const int col_blocks = (boxes_num + threadsPerBlock - 1) / threadsPerBlock;
  const int col_blocks_alloc = GET_BLOCKS(boxes_num, threadsPerBlock);
  Tensor mask =
      at::empty({boxes_num, col_blocks}, boxes.options().dtype(at::kLong));
  dim3 blocks(col_blocks_alloc, col_blocks_alloc);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  nms_cuda<<<blocks, threads, 0, stream>>>(
      boxes_num, iou_threshold, offset, boxes_sorted.data_ptr<float>(),
      (unsigned long long*)mask.data_ptr<int64_t>());

  // Filter the boxes which should be kept.
  at::Tensor keep_t = at::zeros(
      {boxes_num}, boxes.options().dtype(at::kBool).device(at::kCUDA));
  gather_keep_from_mask<<<1, min(col_blocks, THREADS_PER_BLOCK),
                          col_blocks * sizeof(unsigned long long), stream>>>(
      keep_t.data_ptr<bool>(), (unsigned long long*)mask.data_ptr<int64_t>(),
      boxes_num);
  AT_CUDA_CHECK(hipGetLastError());
  return order_t.masked_select(keep_t);
}

}  // namespace ops
}  // namespace mmcv