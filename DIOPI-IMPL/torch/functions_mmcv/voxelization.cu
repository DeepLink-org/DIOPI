#include "hip/hip_runtime.h"
/**
 * @file
 * @author DeepLink
 * @copyright  (c) 2023, DeepLink.
 */

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/library.h>

#include "../cuda_helpers.h"

namespace mmcv {
namespace ops {

using namespace at;

typedef enum { SUM = 0, MEAN = 1, MAX = 2 } reduce_t;

template <typename T, typename T_int>
__global__ void dynamic_voxelize_kernel(
    const T* points, T_int* coors, const float voxel_x, const float voxel_y,
    const float voxel_z, const float coors_x_min, const float coors_y_min,
    const float coors_z_min, const float coors_x_max, const float coors_y_max,
    const float coors_z_max, const int grid_x, const int grid_y,
    const int grid_z, const int num_points, const int num_features,
    const int NDim) {
  //   const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
  CUDA_1D_KERNEL_LOOP(index, num_points) {
    // To save some computation
    auto points_offset = points + index * num_features;
    auto coors_offset = coors + index * NDim;
    int c_x = floorf((points_offset[0] - coors_x_min) / voxel_x);
    if (c_x < 0 || c_x >= grid_x) {
      coors_offset[0] = -1;
      continue;
    }

    int c_y = floorf((points_offset[1] - coors_y_min) / voxel_y);
    if (c_y < 0 || c_y >= grid_y) {
      coors_offset[0] = -1;
      coors_offset[1] = -1;
      continue;
    }

    int c_z = floorf((points_offset[2] - coors_z_min) / voxel_z);
    if (c_z < 0 || c_z >= grid_z) {
      coors_offset[0] = -1;
      coors_offset[1] = -1;
      coors_offset[2] = -1;
    } else {
      coors_offset[0] = c_z;
      coors_offset[1] = c_y;
      coors_offset[2] = c_x;
    }
  }
}

template <typename T, typename T_int>
__global__ void assign_point_to_voxel(const int nthreads, const T* points,
                                      T_int* point_to_voxelidx,
                                      T_int* coor_to_voxelidx, T* voxels,
                                      const int max_points,
                                      const int num_features,
                                      const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    // const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
    int index = thread_idx / num_features;

    int num = point_to_voxelidx[index];
    int voxelidx = coor_to_voxelidx[index];
    if (num > -1 && voxelidx > -1) {
      auto voxels_offset =
          voxels + voxelidx * max_points * num_features + num * num_features;

      int k = thread_idx % num_features;
      voxels_offset[k] = points[thread_idx];
    }
  }
}

template <typename T, typename T_int>
__global__ void assign_voxel_coors(const int nthreads, T_int* coor,
                                   T_int* point_to_voxelidx,
                                   T_int* coor_to_voxelidx, T_int* voxel_coors,
                                   const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    // const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
    // if (index >= num_points) return;
    int index = thread_idx / NDim;
    int num = point_to_voxelidx[index];
    int voxelidx = coor_to_voxelidx[index];
    if (num == 0 && voxelidx > -1) {
      auto coors_offset = voxel_coors + voxelidx * NDim;
      int k = thread_idx % NDim;
      coors_offset[k] = coor[thread_idx];
    }
  }
}

template <typename T_int>
__global__ void point_to_voxelidx_kernel(const T_int* coor,
                                         T_int* point_to_voxelidx,
                                         T_int* point_to_pointidx,
                                         const int max_points,
                                         const int max_voxels,
                                         const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(index, num_points) {
    auto coor_offset = coor + index * NDim;
    // skip invalid points
    if (coor_offset[0] == -1) continue;

    int num = 0;
    int coor_x = coor_offset[0];
    int coor_y = coor_offset[1];
    int coor_z = coor_offset[2];
    // only calculate the coors before this coor[index]
    for (int i = 0; i < index; ++i) {
      auto prev_coor = coor + i * NDim;
      if (prev_coor[0] == -1) continue;

      // Find all previous points that have the same coors
      // if find the same coor, record it
      if ((prev_coor[0] == coor_x) && (prev_coor[1] == coor_y) &&
          (prev_coor[2] == coor_z)) {
        num++;
        if (num == 1) {
          // point to the same coor that first show up
          point_to_pointidx[index] = i;
        } else if (num >= max_points) {
          // out of boundary
          break;
        }
      }
    }
    if (num == 0) {
      point_to_pointidx[index] = index;
    }
    if (num < max_points) {
      point_to_voxelidx[index] = num;
    }
  }
}

template <typename T_int>
__global__ void determin_voxel_num(
    // const T_int* coor,
    T_int* num_points_per_voxel, T_int* point_to_voxelidx,
    T_int* point_to_pointidx, T_int* coor_to_voxelidx, T_int* voxel_num,
    const int max_points, const int max_voxels, const int num_points) {
  // only calculate the coors before this coor[index]
  for (int i = 0; i < num_points; ++i) {
    int point_pos_in_voxel = point_to_voxelidx[i];
    // record voxel
    if (point_pos_in_voxel == -1) {
      // out of max_points or invalid point
      continue;
    } else if (point_pos_in_voxel == 0) {
      // record new voxel
      int voxelidx = voxel_num[0];
      if (voxel_num[0] >= max_voxels) continue;
      voxel_num[0] += 1;
      coor_to_voxelidx[i] = voxelidx;
      num_points_per_voxel[voxelidx] = 1;
    } else {
      int point_idx = point_to_pointidx[i];
      int voxelidx = coor_to_voxelidx[point_idx];
      if (voxelidx != -1) {
        coor_to_voxelidx[i] = voxelidx;
        num_points_per_voxel[voxelidx] += 1;
      }
    }
  }
}

__global__ void nondeterministic_get_assign_pos(
    const int nthreads, const int32_t* coors_map, int32_t* pts_id,
    int32_t* coors_count, int32_t* reduce_count, int32_t* coors_order) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    int coors_idx = coors_map[thread_idx];
    if (coors_idx > -1) {
      int32_t coors_pts_pos = atomicAdd(&reduce_count[coors_idx], 1);
      pts_id[thread_idx] = coors_pts_pos;
      if (coors_pts_pos == 0) {
        coors_order[coors_idx] = atomicAdd(coors_count, 1);
      }
    }
  }
}

template <typename T>
__global__ void nondeterministic_assign_point_voxel(
    const int nthreads, const T* points, const int32_t* coors_map,
    const int32_t* pts_id, const int32_t* coors_in, const int32_t* reduce_count,
    const int32_t* coors_order, T* voxels, int32_t* coors, int32_t* pts_count,
    const int max_voxels, const int max_points, const int num_features,
    const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    int coors_idx = coors_map[thread_idx];
    int coors_pts_pos = pts_id[thread_idx];
    if (coors_idx > -1 && coors_pts_pos < max_points) {
      int coors_pos = coors_order[coors_idx];
      if (coors_pos < max_voxels) {
        auto voxels_offset =
            voxels + (coors_pos * max_points + coors_pts_pos) * num_features;
        auto points_offset = points + thread_idx * num_features;
        for (int k = 0; k < num_features; k++) {
          voxels_offset[k] = points_offset[k];
        }
        if (coors_pts_pos == 0) {
          pts_count[coors_pos] = min(reduce_count[coors_idx], max_points);
          auto coors_offset = coors + coors_pos * NDim;
          auto coors_in_offset = coors_in + coors_idx * NDim;
          for (int k = 0; k < NDim; k++) {
            coors_offset[k] = coors_in_offset[k];
          }
        }
      }
    }
  }
}

int HardVoxelizeForwardCUDAKernelLauncher(
    const at::Tensor &points, at::Tensor &voxels, at::Tensor &coors,
    at::Tensor &num_points_per_voxel, const std::vector<float> voxel_size,
    const std::vector<float> coors_range, const int max_points,
    const int max_voxels, const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device

  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  // map points to voxel coors
  at::Tensor temp_coors =
      at::zeros({num_points, NDim}, points.options().dtype(at::kInt));

  dim3 grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 block(512);

  // 1. link point to corresponding voxel coors
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "hard_voxelize_kernel", ([&] {
        dynamic_voxelize_kernel<scalar_t, int><<<grid, block, 0, stream>>>(
            points.contiguous().data_ptr<scalar_t>(),
            temp_coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
            coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
            coors_z_max, grid_x, grid_y, grid_z, num_points, num_features,
            NDim);
      }));

  AT_CUDA_CHECK(hipGetLastError());

  // 2. map point to the idx of the corresponding voxel, find duplicate coor
  // create some temporary variables
  auto point_to_pointidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto point_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));

  dim3 map_grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 map_block(512);

  AT_DISPATCH_ALL_TYPES(
      temp_coors.scalar_type(), "determin_duplicate", ([&] {
        point_to_voxelidx_kernel<int><<<map_grid, map_block, 0, stream>>>(
            temp_coors.contiguous().data_ptr<int>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            point_to_pointidx.contiguous().data_ptr<int>(), max_points,
            max_voxels, num_points, NDim);
      }));

  AT_CUDA_CHECK(hipGetLastError());

  // 3. determine voxel num and voxel's coor index
  // make the logic in the CUDA device could accelerate about 10 times
  auto coor_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto voxel_num = at::zeros(
      {
          1,
      },
      points.options().dtype(at::kInt));  // must be zero from the beginning

  AT_DISPATCH_ALL_TYPES(temp_coors.scalar_type(), "determin_duplicate", ([&] {
                          determin_voxel_num<int><<<1, 1, 0, stream>>>(
                              num_points_per_voxel.contiguous().data_ptr<int>(),
                              point_to_voxelidx.contiguous().data_ptr<int>(),
                              point_to_pointidx.contiguous().data_ptr<int>(),
                              coor_to_voxelidx.contiguous().data_ptr<int>(),
                              voxel_num.contiguous().data_ptr<int>(),
                              max_points, max_voxels, num_points);
                        }));

  AT_CUDA_CHECK(hipGetLastError());

  // 4. copy point features to voxels
  // Step 4 & 5 could be parallel
  auto pts_output_size = num_points * num_features;
  dim3 cp_grid(std::min(at::cuda::ATenCeilDiv(pts_output_size, 512), 4096));
  dim3 cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_point_to_voxel<float, int><<<cp_grid, cp_block, 0, stream>>>(
            pts_output_size, points.contiguous().data_ptr<float>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            coor_to_voxelidx.contiguous().data_ptr<int>(),
            voxels.contiguous().data_ptr<float>(), max_points, num_features,
            num_points, NDim);
      }));
  //   hipDeviceSynchronize();
  //   AT_CUDA_CHECK(hipGetLastError());

  // 5. copy coors of each voxels
  auto coors_output_size = num_points * NDim;
  dim3 coors_cp_grid(
      std::min(at::cuda::ATenCeilDiv(coors_output_size, 512), 4096));
  dim3 coors_cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_voxel_coors<float, int>
            <<<coors_cp_grid, coors_cp_block, 0, stream>>>(
                coors_output_size, temp_coors.contiguous().data_ptr<int>(),
                point_to_voxelidx.contiguous().data_ptr<int>(),
                coor_to_voxelidx.contiguous().data_ptr<int>(),
                coors.contiguous().data_ptr<int>(), num_points, NDim);
      }));

  AT_CUDA_CHECK(hipGetLastError());

  auto voxel_num_cpu = voxel_num.to(at::kCPU);
  int voxel_num_int = voxel_num_cpu.data_ptr<int>()[0];

  return voxel_num_int;
}

int NondeterministicHardVoxelizeForwardCUDAKernelLauncher(
    const at::Tensor &points, at::Tensor &voxels, at::Tensor &coors,
    at::Tensor &num_points_per_voxel, const std::vector<float> voxel_size,
    const std::vector<float> coors_range, const int max_points,
    const int max_voxels, const int NDim = 3) {
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  if (num_points == 0) return 0;

  dim3 blocks(
      std::min(at::cuda::ATenCeilDiv(num_points, THREADS_PER_BLOCK), 4096));
  dim3 threads(THREADS_PER_BLOCK);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  // map points to voxel coors
  at::Tensor temp_coors =
      at::zeros({num_points, NDim}, points.options().dtype(at::kInt));

  // 1. link point to corresponding voxel coors
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "hard_voxelize_kernel", ([&] {
        dynamic_voxelize_kernel<scalar_t, int><<<blocks, threads, 0, stream>>>(
            points.contiguous().data_ptr<scalar_t>(),
            temp_coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
            coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
            coors_z_max, grid_x, grid_y, grid_z, num_points, num_features,
            NDim);
      }));

  at::Tensor coors_map;
  at::Tensor reduce_count;

  auto coors_clean = temp_coors.masked_fill(temp_coors.lt(0).any(-1, true), -1);

  std::tie(temp_coors, coors_map, reduce_count) =
      at::unique_dim(coors_clean, 0, true, true, false);

  if (temp_coors[0][0].lt(0).item<bool>()) {
    // the first element of temp_coors is (-1,-1,-1) and should be removed
    temp_coors = temp_coors.slice(0, 1);
    coors_map = coors_map - 1;
  }

  int num_coors = temp_coors.size(0);
  temp_coors = temp_coors.to(at::kInt);
  coors_map = coors_map.to(at::kInt);

  at::Tensor coors_count = at::zeros({1}, coors_map.options());
  at::Tensor coors_order = at::empty({num_coors}, coors_map.options());
  at::Tensor pts_id = at::zeros({num_points}, coors_map.options());
  reduce_count = at::zeros({num_coors}, coors_map.options());

  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "get_assign_pos", ([&] {
        nondeterministic_get_assign_pos<<<blocks, threads, 0, stream>>>(
            num_points, coors_map.contiguous().data_ptr<int32_t>(),
            pts_id.contiguous().data_ptr<int32_t>(),
            coors_count.contiguous().data_ptr<int32_t>(),
            reduce_count.contiguous().data_ptr<int32_t>(),
            coors_order.contiguous().data_ptr<int32_t>());
      }));

  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        nondeterministic_assign_point_voxel<scalar_t>
            <<<blocks, threads, 0, stream>>>(
                num_points, points.contiguous().data_ptr<scalar_t>(),
                coors_map.contiguous().data_ptr<int32_t>(),
                pts_id.contiguous().data_ptr<int32_t>(),
                temp_coors.contiguous().data_ptr<int32_t>(),
                reduce_count.contiguous().data_ptr<int32_t>(),
                coors_order.contiguous().data_ptr<int32_t>(),
                voxels.contiguous().data_ptr<scalar_t>(),
                coors.contiguous().data_ptr<int32_t>(),
                num_points_per_voxel.contiguous().data_ptr<int32_t>(),
                max_voxels, max_points, num_features, NDim);
      }));
  AT_CUDA_CHECK(hipGetLastError());
  return max_voxels < num_coors ? max_voxels : num_coors;
}

void DynamicVoxelizeForwardCUDAKernelLauncher(
    const at::Tensor &points, at::Tensor &coors,
    const std::vector<float> voxel_size, const std::vector<float> coors_range,
    const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device

  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  const int col_blocks = at::cuda::ATenCeilDiv(num_points, THREADS_PER_BLOCK);
  dim3 blocks(col_blocks);
  dim3 threads(THREADS_PER_BLOCK);

  AT_DISPATCH_ALL_TYPES(points.scalar_type(), "dynamic_voxelize_kernel", [&] {
    dynamic_voxelize_kernel<scalar_t, int><<<blocks, threads, 0, stream>>>(
        points.contiguous().data_ptr<scalar_t>(),
        coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
        coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
        coors_z_max, grid_x, grid_y, grid_z, num_points, num_features, NDim);
  });

  AT_CUDA_CHECK(hipGetLastError());
}

}  // namespace ops
}  // namespace mmcv